#include "hip/hip_runtime.h"
/*
 * Author: Rachel
 * <zhangruiqing01@baidu.com>
 *
 * File: bitonic_sort.cu
 * Create Date: 2015-08-05 17:10:44
 *
 */
 
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include"gputimer.h"
#include<time.h>
#define NThreads 8
#define NBlocks 4
 
#define Num NThreads*NBlocks
 
using namespace Gadgetron;
 
 
__device__ void swap(int &a, int &b){
    int t = a;
    a = b;
    b = t;
}
 
__global__ void bitonic_sort(int* arr){
    extern __shared__ int shared_arr[];
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //const unsigned int tid = threadIdx.x;
    shared_arr[tid] = arr[tid];
    __syncthreads();
 
    //for(int i=2; i<=blociDim.x; i<<=1){
    for(unsigned int i=2; i<=Num; i<<=1){
        for(unsigned int j=i>>1; j>0; j>>=1){
            unsigned int tid_comp = tid ^ j;
            if(tid_comp > tid){
                if((tid & i)==0){ //ascending
                    if(shared_arr[tid]>shared_arr[tid_comp]){
                        swap(shared_arr[tid],shared_arr[tid_comp]);
                    }
                }
                else{ //desending
                    if(shared_arr[tid]<shared_arr[tid_comp]){
                        swap(shared_arr[tid],shared_arr[tid_comp]);
                    }
                }
            }
            __syncthreads();
        }
    }
    arr[tid] = shared_arr[tid];
}
 
int main(int argc, char* argv[])
{
    GPUTimer timer;
    int* arr= (int*) malloc(Num*sizeof(int));
 
    //init array value
    time_t t;
    srand((unsigned)time(&t));
    for(int i=0;i<Num;i++){
        arr[i] = rand() % 1000; 
    }
 
    //init device variable
    int* ptr;
    hipMalloc((void**)&ptr,Num*sizeof(int));
    hipMemcpy(ptr,arr,Num*sizeof(int),hipMemcpyHostToDevice);
 
    for(int i=0;i<Num;i++){
        printf("%d\t",arr[i]);
    }
    printf("\n");
 
 
    dim3 blocks(NBlocks,1);
    dim3 threads(NThreads,1);
 
    timer.start();
    bitonic_sort<<<blocks,threads,Num*sizeof(int)>>>(ptr);
    //bitonic_sort<<<1,Num,Num*sizeof(int)>>>(ptr);
    timer.stop();
 
 
    hipMemcpy(arr,ptr,Num*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<Num;i++){
        printf("%d\t",arr[i]);
    }
    printf("\n");
 
    hipFree(ptr);
    return 0;
}
