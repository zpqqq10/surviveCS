
#include <hip/hip_runtime.h>
__global__ void global_reduce_kernel(float *d_out, float *d_in){
    int myID = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    // reduction
    for (unsigned int s = blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            d_in[myID] += d_in[myID + s];
        }
        __syncthreads();
    }
    
    // only thread 0 writes result for this block back to global memory
    if (tid == 0)
        d_out[blockIdx.x] = d_in[myID];
}

__global__ void shmem_reduce_kernel(float *d_out, float *d_in){
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myID = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    // load shared memory from global memory
    sdata[tid] = d_in[myID]; 
    __syncthreads();

    // reduction
    for (unsigned int s = blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            sdata[myID] += sdata[myID + s];
        }
        __syncthreads();
    }
    
    // only thread 0 writes result for this block back to global memory
    if (tid == 0)
        d_out[blockIdx.x] = sdata[myID];
}