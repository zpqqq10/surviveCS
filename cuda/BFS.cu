#include "hip/hip_runtime.h"
// launch v threads
__global__ void initialize_vertices(Vertex *vertices, int startting_vertex, int num_vertices){
    int v = threadIdx.x + blockDim.x * blockIdx.x; 
    if(v == starting_vertex){
        vertices[v] = 0; 
    }
    else {
        vertices[v] = -1;
    }
}

// launch e threads
__global__ void bfs(const Edge *edges, Vertex *vertices, int current_depth){
    int e = threadIdx.x + blockDim.x * blockIdx.x; 
    int vfirst = edges[e].first;    // the vertex on the one end
    int dfirst = vertices[vfirst];  // the distance of one vertex
    int vsecond = edges[e].second;  // the vertex on the other end
    int dsecond = vertices[vsecond]; // the distance of another vertex

    if((dfirst == current_depth) && (dsecond == -1)){
        // the first vertex is visited but the second one is not 
        vertices[vsecond] = dfirst + 1;
        done = false;   // global
    }
    if((dsecond == current_depth) && (dfirst == -1)){
        // the second vertex is visited but the first one is not 
        vertices[first] = dsecond + 1;
        done = false;
    }
    // do nothing is both are visited or unvisited
}

void main(){
    ...
    // initialize d_done to be true
    while(!h_done){
        bfs(edges, vertices);
        hipMemcpy(&h_done, &d_done, sizeof(bool), cudaDeviceToHost);
    }
    ...
}