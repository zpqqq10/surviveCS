#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void square(float *d_out, float *d_in)
{
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char **argv){
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array
    float h_in[ARRAY_SIZE]; 
    for (int i = 0; i < ARRAY_SIZE; i++){
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float *d_in, *d_out; 

    // allocate GPU memory
    hipMalloc((void **)&d_in, ARRAY_BYTES); 
    hipMalloc((void **)&d_out, ARRAY_BYTES); 

    // transfer the array to GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, cudaMemcpyHostToDevic);

    // launch the kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print
    
    // free memory
    hipFree(d_in); 
    hipFree(d_out);
}