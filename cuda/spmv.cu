
#include <hip/hip_runtime.h>
__global__ void spmv_csr_scalar_kernel(
    const int num_rows, const int *rowptr,
    const int *index, const float *value,
    const float *vec, // operand vector
    float *y)   // destination vector
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    if(row < num_rows){
        float dot = 0; 
        int row_start = rowptr[row];
        int row_end = rowptr[row+1]; 
        for (int j = row_start; j < row_end; j++){
            dot += value[j] * vec[index[j]];
        }
        y[row] += dot; 
    }
}