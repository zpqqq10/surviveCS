#include "hip/hip_runtime.h"
__device__ float3 tile_calculation(Params myParams, float3 force){
    int i; 
    extern __share__ Params[] sourceParams;
    for (i = 0; i < blockDim.x; i++)
        force += bodyBodyInteraction(myParams, sourceParams[i]);
    return force;
}