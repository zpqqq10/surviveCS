#include "hip/hip_runtime.h"
#include<stdio.h>
#include "gputimer.h"

// matrix size
const int N = 1024;
// tile size
const int K = 16;
// serial in CPU
void transpose_CPU(float in[], float out[]){
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++)
            out[j + i*N] = in[i + j*N];
}

// serial in GPU
// <<<1, 1>>>
__global__ void transpose_serial(float in[], float out[]){
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++)
            out[j + i*N] = in[i + j*N];
}

// parallel in GPU
// <<<1, N>>>
__global__ void transpose_parallel_per_row(float in[], float out[]){
    int i = threadIdx.x;
    for (int j = 0; j < N; j++)
            out[j + i*N] = in[i + j*N];
}

// parallel in GPU by zpq
// <<<N, N>>>
__global__ void transpose_parallel_per_element(float in[], float out[]){
    int i = threadIdx.x;
    int j = blockIdx.x; 
    // blockDim.x == N
    out[j + i*N] = in[i + j*N];
}

// parallel in GPU
// to be launched with one thread per element, in KxK threadblocks
// thread (x, y) in grid writes element (i, j) of output matrix
// given that a block can contain 1024 threads at most, it is advised to use KxK blocks
/***************
 * dim3 block(N/K, N/K); 
 * dim3 threads(K, K); 
 * <<<blocks, threads>>>
**********/
__global__ void transpose_parallel_per_element(float in[], float out[]){
    int i = blockIdx.x * K + threadIdx.x;
    int j = blockIdx.y * K + threadIdx.y;
    out[j + i*N] = in[i + j*N];
}


// K = 16 will be faster than K = 32
// to be launched with one thread per element in KxK threadblocks
// thread blocks read & write tiles in coalesced fashion
// adjacent threads read adjacent input elements and write adjacent output elements
__global__ void transpose_parallel_per_element_tiled(float in[], float out[]){
    // (i, j) locations of the top left tile corners for input & output matrices
    int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
    int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;
    // for readability
    int x = threadIdx.x;
    int y = threadIdx.y;

    __shared__ float tile[K][K]; 

    // coalesced read from global memory, transposed and write into shared memory
    tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * N];
    __syncthreads();    // cause huge latency
    // read from shared memory and coalesced write to global memory
    out[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}