#include "hip/hip_runtime.h"
__global__ void quicksort(int *data, int left, int right){
    int nleft, nright; 
    hipStream_t s1, s2; 
    partition(data+left, data+right, data[left], nleft, nright); 

    if(left < nright){
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort<<< ..., s1 >>>(data, left, nright); 
    }
    if(nleft < right){
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort<<< ..., s2 >>>(data, nleft, right); 
    }
}